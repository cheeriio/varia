
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <iostream> 
#include <sys/time.h>
#include <hip/hip_runtime_api.h>

using namespace std;
using real = float;

#define ASSERT_CUDA(expr)                                    \
do {                                                         \
    hipError_t status = (expr);                             \
    if (status != hipSuccess) {                             \
        printf("%s\n", hipGetErrorString(status));          \
        exit(1);                                             \
    }                                                        \
} while(0)  

const real MAX_VAL = 100.0;
const int NR_OF_EXPERIMENTS = 20;

int N;

void matrix_generate(real* A);

void gpu_matrix_multiply_v1_8(real* A, real* B, real* C);
void gpu_matrix_multiply_v1_16(real* A, real* B, real* C);
void gpu_matrix_multiply_v1_32(real* A, real* B, real* C);

void gpu_matrix_multiply_v2_32(real* A, real* B, real* C);
void gpu_matrix_multiply_v2_64(real* A, real* B, real* C);
void gpu_matrix_multiply_v2_96(real* A, real* B, real* C);

void gpu_matrix_multiply_v3_32(real* A, real* B, real* C);
void gpu_matrix_multiply_v3_64(real* A, real* B, real* C);

void gpu_matrix_multiply_v4_32(real* A, real* B, real* C);

void gpu_matrix_multiply_v5_32(real* A, real* B, real* C);

void cpu_matrix_multiply(real* A, real* B, real* C);


int main(int argc, char **argv){
    if(argc != 2){
        cout << "Wywołanie: " << argv[0] << " N" << endl;
        return 0;
    }
    N = atoi(argv[1]);

    real *A, *B, *C_v1_8, *C_v1_16, *C_v1_32, *C_v2_32, *C_v2_64, *C_v2_96, *C_v3_32, *C_v3_64, *C_v4_32, *C_v5_32, *C_cpu;
    A = (real*)malloc(sizeof(real) * N * N);
    B = (real*)malloc(sizeof(real) * N * N);
    C_v1_8 = (real*)malloc(sizeof(real) * N * N);
    C_v1_16 = (real*)malloc(sizeof(real) * N * N);
    C_v1_32 = (real*)malloc(sizeof(real) * N * N);
    C_v2_32 = (real*)malloc(sizeof(real) * N * N);
    C_v2_64 = (real*)malloc(sizeof(real) * N * N);
    C_v2_96 = (real*)malloc(sizeof(real) * N * N);
    C_v3_32 = (real*)malloc(sizeof(real) * N * N);
    C_v3_64 = (real*)malloc(sizeof(real) * N * N);
    C_v4_32 = (real*)malloc(sizeof(real) * N * N);
    C_v5_32 = (real*)malloc(sizeof(real) * N * N);
    C_cpu = (real*)malloc(sizeof(real) * N * N);

    srand(777);
    matrix_generate(A);
    matrix_generate(B);

    time_t start, end;
    int dif_elems = 0;
    real deviation = 0;
    printf("id,cpu,gpu_v1_8,dif_elems,deviation,gpu_v1_16,dif_elems,deviation,gpu_v1_32,dif_elems,deviation,"
            "gpu_v2_32,dif_elems,deviation,gpu_v2_64,dif_elems,deviation,gpu_v2_64,dif_elems,deviation,"
            "gpu_v3_32,dif_elems,deviation,gpu_v3_64,dif_elems,deviation,gpu_v4_32,dif_elems,deviation,gpu_v5_32,dif_elems,deviation\n");
    for(int exp = 0; exp < NR_OF_EXPERIMENTS; exp++){
        printf("%d,", exp + 1);

        // CPU
        start = clock();
        cpu_matrix_multiply(A, B, C_cpu);
        end = clock();
        printf("%lf,", 1.0*(end-start)/CLOCKS_PER_SEC);

        // GPU_v1 8x8
        start = clock();
        gpu_matrix_multiply_v1_8(A, B, C_v1_8);
        end = clock();
        printf("%lf,", 1.0*(end-start)/CLOCKS_PER_SEC);
        for(int i = 0; i < N; i++){
            for(int j = 0; j < N; j++){
                if(C_v1_8[i*N + j] != C_cpu[i*N + j]){
                    dif_elems++;
                    deviation += abs(C_v1_8[i*N + j] - C_cpu[i*N + j]);
                }
            }
        }
        printf("%d,%lf,", dif_elems, deviation);
        dif_elems = 0;
        deviation = 0.0;

        // GPU_v1 16x16
        start = clock();
        gpu_matrix_multiply_v1_16(A, B, C_v1_16);
        end = clock();
        printf("%lf,", 1.0*(end-start)/CLOCKS_PER_SEC);
        for(int i = 0; i < N; i++){
            for(int j = 0; j < N; j++){
                if(C_v1_16[i*N + j] != C_cpu[i*N + j]){
                    dif_elems++;
                    deviation += abs(C_v1_16[i*N + j] - C_cpu[i*N + j]);
                }
            }
        }
        printf("%d,%lf,", dif_elems, deviation);
        dif_elems = 0;
        deviation = 0.0;

        // GPU_v1 32x32
        start = clock();
        gpu_matrix_multiply_v1_32(A, B, C_v1_32);
        end = clock();
        printf("%lf,", 1.0*(end-start)/CLOCKS_PER_SEC);
        for(int i = 0; i < N; i++){
            for(int j = 0; j < N; j++){
                if(C_v1_32[i*N + j] != C_cpu[i*N + j]){
                    dif_elems++;
                    deviation += abs(C_v1_32[i*N + j] - C_cpu[i*N + j]);
                }
            }
        }
        printf("%d,%lf,", dif_elems, deviation);
        dif_elems = 0;
        deviation = 0.0;
        
        // GPU_v2 32x1
        start = clock();
        gpu_matrix_multiply_v2_32(A, B, C_v2_32);
        end = clock();
        printf("%lf,", 1.0*(end-start)/CLOCKS_PER_SEC);
        for(int i = 0; i < N; i++){
            for(int j = 0; j < N; j++){
                if(C_v2_32[i*N + j] != C_cpu[i*N + j]){
                    dif_elems++;
                    deviation += abs(C_v2_32[i*N + j] - C_cpu[i*N + j]);
                }
            }
        }
        printf("%d,%lf,", dif_elems, deviation);
        dif_elems = 0;
        deviation = 0.0;

        // GPU_v2 64x1
        start = clock();
        gpu_matrix_multiply_v2_64(A, B, C_v2_64);
        end = clock();
        printf("%lf,", 1.0*(end-start)/CLOCKS_PER_SEC);
        for(int i = 0; i < N; i++){
            for(int j = 0; j < N; j++){
                if(C_v2_64[i*N + j] != C_cpu[i*N + j]){
                    dif_elems++;
                    deviation += abs(C_v2_64[i*N + j] - C_cpu[i*N + j]);
                }
            }
        }
        printf("%d,%lf,", dif_elems, deviation);
        dif_elems = 0;
        deviation = 0.0;

        // GPU_v2 96x1
        start = clock();
        gpu_matrix_multiply_v2_96(A, B, C_v2_96);
        end = clock();
        printf("%lf,", 1.0*(end-start)/CLOCKS_PER_SEC);
        for(int i = 0; i < N; i++){
            for(int j = 0; j < N; j++){
                if(C_v2_96[i*N + j] != C_cpu[i*N + j]){
                    dif_elems++;
                    deviation += abs(C_v2_96[i*N + j] - C_cpu[i*N + j]);
                }
            }
        }
        printf("%d,%lf,", dif_elems, deviation);
        dif_elems = 0;
        deviation = 0.0;

        // GPU_v3 32x1
        start = clock();
        gpu_matrix_multiply_v3_32(A, B, C_v3_32);
        end = clock();
        printf("%lf,", 1.0*(end-start)/CLOCKS_PER_SEC);
        for(int i = 0; i < N; i++){
            for(int j = 0; j < N; j++){
                if(C_v3_32[i*N + j] != C_cpu[i*N + j]){
                    dif_elems++;
                    deviation += abs(C_v3_32[i*N + j] - C_cpu[i*N + j]);
                }
            }
        }
        printf("%d,%lf,", dif_elems, deviation);
        dif_elems = 0;
        deviation = 0.0;
        
        // GPU_v3 64x1
        start = clock();
        gpu_matrix_multiply_v3_64(A, B, C_v3_64);
        end = clock();
        printf("%lf,", 1.0*(end-start)/CLOCKS_PER_SEC);
        for(int i = 0; i < N; i++){
            for(int j = 0; j < N; j++){
                if(C_v3_64[i*N + j] != C_cpu[i*N + j]){
                    dif_elems++;
                    deviation += abs(C_v3_64[i*N + j] - C_cpu[i*N + j]);
                }
            }
        }
        printf("%d,%lf,", dif_elems, deviation);
        dif_elems = 0;
        deviation = 0.0;
        
        // GPU_v4 32x1
        start = clock();
        gpu_matrix_multiply_v4_32(A, B, C_v4_32);
        end = clock();
        printf("%lf,", 1.0*(end-start)/CLOCKS_PER_SEC);
        for(int i = 0; i < N; i++){
            for(int j = 0; j < N; j++){
                if(C_v4_32[i*N + j] != C_cpu[i*N + j]){
                    dif_elems++;
                    deviation += abs(C_v4_32[i*N + j] - C_cpu[i*N + j]);
                }
            }
        }
        printf("%d,%lf,", dif_elems, deviation);
        dif_elems = 0;
        deviation = 0.0;

        // GPU_v5 32x1
        start = clock();
        gpu_matrix_multiply_v5_32(A, B, C_v5_32);
        end = clock();
        printf("%lf,", 1.0*(end-start)/CLOCKS_PER_SEC);
        for(int i = 0; i < N; i++){
            for(int j = 0; j < N; j++){
                if(C_v5_32[i*N + j] != C_cpu[i*N + j]){
                    dif_elems++;
                    deviation += abs(C_v5_32[i*N + j] - C_cpu[i*N + j]);
                }
            }
        }
        printf("%d,%lf\n", dif_elems, deviation);
        dif_elems = 0;
        deviation = 0.0;
    }
    return 0;
}

void matrix_generate(real* A){
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            float r = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
            r = (r - 0.5) * 2;  // r is in [-1, 1]
            A[i*N + j] = r * MAX_VAL;
        }
    }
}

inline void make_gpu_matrices(real*& A_gpu, real*& B_gpu, real*& C_gpu, real* A, real* B){
    ASSERT_CUDA(hipMalloc((void**)&A_gpu, sizeof(real) * N * N));
    ASSERT_CUDA(hipMalloc((void**)&B_gpu, sizeof(real) * N * N));
    ASSERT_CUDA(hipMalloc((void**)&C_gpu, sizeof(real) * N * N));

    ASSERT_CUDA(hipMemcpy(A_gpu, A, sizeof(real) * N * N, hipMemcpyHostToDevice));
    ASSERT_CUDA(hipMemcpy(B_gpu, B, sizeof(real) * N * N, hipMemcpyHostToDevice));
}

inline void delete_gpu_matrices(real* A_gpu, real* B_gpu, real* C_gpu, real* C){
    ASSERT_CUDA(hipMemcpy(C, C_gpu, sizeof(real) * N * N, hipMemcpyDeviceToHost));

    ASSERT_CUDA(hipFree(A_gpu));
    ASSERT_CUDA(hipFree(B_gpu));
    ASSERT_CUDA(hipFree(C_gpu));
}

// GPU_v1

__global__ void kernel_v1(real* A, real* B, real* C, int N){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if(i >= N || j >= N)
        return;

    C[i*N + j] = 0;
    for(int k = 0; k < N; k++)
        C[i*N + j] += A[i*N + k] * B[k*N + j];
}

void gpu_matrix_multiply_v1_8(real* A, real* B, real* C){
    real *A_gpu = NULL, *B_gpu = NULL, *C_gpu = NULL;
    make_gpu_matrices(A_gpu, B_gpu, C_gpu, A, B);
    dim3 threadsPerBlock(8, 8, 1);
    dim3 numBlocks(N / 8 + 1, N / 8 + 1, 1);
    kernel_v1<<<numBlocks, threadsPerBlock, 0>>>(A_gpu, B_gpu, C_gpu, N);
    delete_gpu_matrices(A_gpu, B_gpu, C_gpu, C);
}

void gpu_matrix_multiply_v1_16(real* A, real* B, real* C){
    real *A_gpu = NULL, *B_gpu = NULL, *C_gpu = NULL;
    make_gpu_matrices(A_gpu, B_gpu, C_gpu, A, B);
    dim3 threadsPerBlock(16, 16, 1);
    dim3 numBlocks(N / 16 + 1, N / 16 + 1, 1);
    kernel_v1<<<numBlocks, threadsPerBlock, 0>>>(A_gpu, B_gpu, C_gpu, N);
    delete_gpu_matrices(A_gpu, B_gpu, C_gpu, C);
}

void gpu_matrix_multiply_v1_32(real* A, real* B, real* C){
    real *A_gpu = NULL, *B_gpu = NULL, *C_gpu = NULL;
    make_gpu_matrices(A_gpu, B_gpu, C_gpu, A, B);
    dim3 threadsPerBlock(32, 32, 1);
    dim3 numBlocks(N / 32 + 1, N / 32 + 1, 1);
    kernel_v1<<<numBlocks, threadsPerBlock, 0>>>(A_gpu, B_gpu, C_gpu, N);
    delete_gpu_matrices(A_gpu, B_gpu, C_gpu, C);
}

// GPU_v2

__global__ void kernel_v2_32(real* A, real* B, real* C, int N){
    __shared__ real result[32 * 32];
    __shared__ real bvec[32];
    int x0 = blockIdx.x * 32;
    int y0 = blockIdx.y * 32;
    int x = x0 + threadIdx.x;
    int y = y0 + threadIdx.x;

    for (int i = 0; i < 32; i++) {
        result[threadIdx.x * 32 + i] = 0;
    }

    for (int i = 0; i < N; i++) {
        // Set the value of currently needed B element in shared memory.
        if(x < N) {
            bvec[threadIdx.x] = B[x + (N * i)];
        } else {
            bvec[threadIdx.x] = 0;
        }

        // Get value of A element currently needed.
        real a;
        if(y < N) {
            a = A[y * N + i];
        } else {
            a = 0;
        }

        // Sync so all bvec values are set.
        __syncthreads();
        for (int j = 0; j < 32; j++) {
            result[threadIdx.x * 32 + j] += a * bvec[j];
        }
        // Sync so every thread gets to finish the job with proper bvec values.
        __syncthreads();
    }

    // Copy result to C array.
    if(y < N) {
        for (int i = 0; i < 32; i++) {
            if(x0 + i < N) {
                C[y * N + x0 + i] = result[threadIdx.x * 32 + i];
            } else
                break;
        }
    }
}

void gpu_matrix_multiply_v2_32(real* A, real* B, real* C){
    real *A_gpu = NULL, *B_gpu = NULL, *C_gpu = NULL;
    make_gpu_matrices(A_gpu, B_gpu, C_gpu, A, B);
    dim3 threadsPerBlock(32, 1, 1);
    dim3 numBlocks(N / 32 + 1, N / 32 + 1, 1);
    kernel_v2_32<<<numBlocks, threadsPerBlock, 0>>>(A_gpu, B_gpu, C_gpu, N);
    delete_gpu_matrices(A_gpu, B_gpu, C_gpu, C);
}

__global__ void kernel_v2_64(real* A, real* B, real* C, int N){
    __shared__ real result[64 * 64];
    __shared__ real bvec[64];
    int x0 = blockIdx.x * 64;
    int y0 = blockIdx.y * 64;
    int x = x0 + threadIdx.x;
    int y = y0 + threadIdx.x;

    for (int i = 0; i < 64; i++) {
        result[threadIdx.x * 64 + i] = 0;
    }

    for (int i = 0; i < N; i++) {
        if(x < N) {
            bvec[threadIdx.x] = B[x + (N * i)];
        } else {
            bvec[threadIdx.x] = 0;
        }

        real a;
        if(y < N) {
            a = A[y * N + i];
        } else {
            a = 0;
        }

        __syncthreads();
        for (int j = 0; j < 64; j++) {
            result[threadIdx.x * 64 + j] += a * bvec[j];
        }
        __syncthreads();
    }

    if(y < N) {
        for (int i = 0; i < 64; i++) {
            if(x0 + i < N) {
                C[y * N + x0 + i] = result[threadIdx.x * 64 + i];
            } else
                break;
        }
    }
}

void gpu_matrix_multiply_v2_64(real* A, real* B, real* C){
    real *A_gpu = NULL, *B_gpu = NULL, *C_gpu = NULL;
    make_gpu_matrices(A_gpu, B_gpu, C_gpu, A, B);
    dim3 threadsPerBlock(64, 1, 1);
    dim3 numBlocks(N / 64 + 1, N / 64 + 1, 1);
    kernel_v2_64<<<numBlocks, threadsPerBlock, 0>>>(A_gpu, B_gpu, C_gpu, N);
    delete_gpu_matrices(A_gpu, B_gpu, C_gpu, C);
}

__global__ void kernel_v2_96(real* A, real* B, real* C, int N){
    __shared__ real result[96 * 96];
    __shared__ real bvec[96];
    int x0 = blockIdx.x * 96;
    int y0 = blockIdx.y * 96;
    int x = x0 + threadIdx.x;
    int y = y0 + threadIdx.x;

    for (int i = 0; i < 96; i++) {
        result[threadIdx.x * 96 + i] = 0;
    }

    for (int i = 0; i < N; i++) {
        if(x < N) {
            bvec[threadIdx.x] = B[x + (N * i)];
        } else {
            bvec[threadIdx.x] = 0;
        }

        real a;
        if(y < N) {
            a = A[y * N + i];
        } else {
            a = 0;
        }

        __syncthreads();
        for (int j = 0; j < 96; j++) {
            result[threadIdx.x * 96 + j] += a * bvec[j];
        }
        __syncthreads();
    }

    if(y < N) {
        for (int i = 0; i < 96; i++) {
            if(x0 + i < N) {
                C[y * N + x0 + i] = result[threadIdx.x * 96 + i];
            } else
                break;
        }
    }
}

void gpu_matrix_multiply_v2_96(real* A, real* B, real* C){
    real *A_gpu = NULL, *B_gpu = NULL, *C_gpu = NULL;
    make_gpu_matrices(A_gpu, B_gpu, C_gpu, A, B);
    dim3 threadsPerBlock(96, 1, 1);
    dim3 numBlocks(N / 96 + 1, N / 96 + 1, 1);
    kernel_v2_96<<<numBlocks, threadsPerBlock, 0>>>(A_gpu, B_gpu, C_gpu, N);
    delete_gpu_matrices(A_gpu, B_gpu, C_gpu, C);
}

// GPU_v3

__global__ void kernel_v3_32(real* A, real* B, real* C, int N){
    __shared__ real result[32 * 32];
    __shared__ real bvec[32];
    int x0 = blockIdx.x * 32;
    int y0 = blockIdx.y * 32;
    int x = x0 + threadIdx.x;
    int y = y0 + threadIdx.x;

    for (int i = 0; i < 32; i++) {
        result[threadIdx.x * 32 + i] = 0;
    }

    for (int i = 0; i < N; i++) {
        if(x < N) {
            bvec[threadIdx.x] = B[x + (N * i)];
        } else {
            bvec[threadIdx.x] = 0;
        }

        real a;
        if(y < N) {
            a = A[y * N + i];
        } else {
            a = 0;
        }

        __syncthreads();
        // Resolve conflicts by adding offset (threadIdx.x). Now no two threads access same memory address.
        for (int j = 0; j < 32; j++) {
            result[threadIdx.x * 32 + (j + threadIdx.x) % 32] += a * bvec[(j + threadIdx.x) % 32];
        }
        __syncthreads();
    }

    if(y < N) {
        for (int i = 0; i < 32; i++) {
            if(x0 + i < N) {
                C[y * N + x0 + i] = result[threadIdx.x * 32 + i];
            } else
                break;
        }
    }
}

void gpu_matrix_multiply_v3_32(real* A, real* B, real* C){
    real *A_gpu = NULL, *B_gpu = NULL, *C_gpu = NULL;
    make_gpu_matrices(A_gpu, B_gpu, C_gpu, A, B);
    dim3 threadsPerBlock(32, 1, 1);
    dim3 numBlocks(N / 32 + 1, N / 32 + 1, 1);
    kernel_v3_32<<<numBlocks, threadsPerBlock, 0>>>(A_gpu, B_gpu, C_gpu, N);
    delete_gpu_matrices(A_gpu, B_gpu, C_gpu, C);
}

__global__ void kernel_v3_64(real* A, real* B, real* C, int N){
    __shared__ real result[64 * 64];
    __shared__ real bvec[64];
    int x0 = blockIdx.x * 64;
    int y0 = blockIdx.y * 64;
    int x = x0 + threadIdx.x;
    int y = y0 + threadIdx.x;

    for (int i = 0; i < 64; i++) {
        result[threadIdx.x * 64 + i] = 0;
    }

    for (int i = 0; i < N; i++) {
        if(x < N) {
            bvec[threadIdx.x] = B[x + (N * i)];
        } else {
            bvec[threadIdx.x] = 0;
        }

        real a;
        if(y < N) {
            a = A[y * N + i];
        } else {
            a = 0;
        }

        __syncthreads();
        for (int j = 0; j < 64; j++) {
            result[threadIdx.x * 64 + (j + threadIdx.x) % 64] += a * bvec[(j + threadIdx.x) % 64];
        }
        __syncthreads();
    }

    if(y < N) {
        for (int i = 0; i < 64; i++) {
            if(x0 + i < N) {
                C[y * N + x0 + i] = result[threadIdx.x * 64 + i];
            } else
                break;
        }
    }
}

void gpu_matrix_multiply_v3_64(real* A, real* B, real* C){
    real *A_gpu = NULL, *B_gpu = NULL, *C_gpu = NULL;
    make_gpu_matrices(A_gpu, B_gpu, C_gpu, A, B);
    dim3 threadsPerBlock(64, 1, 1);
    dim3 numBlocks(N / 64 + 1, N / 64 + 1, 1);
    kernel_v3_64<<<numBlocks, threadsPerBlock, 0>>>(A_gpu, B_gpu, C_gpu, N);
    delete_gpu_matrices(A_gpu, B_gpu, C_gpu, C);
}

// GPU_v4

__global__ void kernel_v4_32(real* A, real* B, real* C, int N){
    __shared__ real result[32 * 32];
    int x0 = blockIdx.x * 32;
    int y0 = blockIdx.y * 32;
    int x = x0 + threadIdx.x;
    int y = y0 + threadIdx.x;

    for (int i = 0; i < 32; i++) {
        result[threadIdx.x * 32 + i] = 0;
    }

    for (int i = 0; i < N; i++) {
        real b;
        if(x < N) {
            b = B[x + (N * i)];
        } else {
            b = 0;
        }

        real a;
        if(y < N) {
            a = A[y * N + i];
        } else {
            a = 0;
        }

        __syncthreads();
        // We take value from thread 'j' using by shuffling.
        for (int j = 0; j < 32; j++) {
            result[threadIdx.x * 32 + (j + threadIdx.x) % 32] += a * __shfl_sync(0xffffffff, b, j);
        }
        __syncthreads();
    }

    if(y < N) {
        for (int i = 0; i < 32; i++) {
            if(x0 + i < N) {
                C[y * N + x0 + i] = result[threadIdx.x * 32 + i];
            } else
                break;
        }
    }
}

void gpu_matrix_multiply_v4_32(real* A, real* B, real* C){
    real *A_gpu = NULL, *B_gpu = NULL, *C_gpu = NULL;
    make_gpu_matrices(A_gpu, B_gpu, C_gpu, A, B);
    dim3 threadsPerBlock(32, 1, 1);
    dim3 numBlocks(N / 32 + 1, N / 32 + 1, 1);
    kernel_v4_32<<<numBlocks, threadsPerBlock, 0>>>(A_gpu, B_gpu, C_gpu, N);
    delete_gpu_matrices(A_gpu, B_gpu, C_gpu, C);
}

// GPU_v5

__global__ void kernel_v5_32(real* A, real* B, real* C, int N){
    // We only store our result row.
    real reg[32];
    int x0 = blockIdx.x * 32;
    int y0 = blockIdx.y * 32;
    int x = x0 + threadIdx.x;
    int y = y0 + threadIdx.x;

    for(int i = 0; i < 32; i++)
        reg[i] = 0;

    for (int i = 0; i < N; i++) {
        real b;
        if(x < N) {
            b = B[x + (N * i)];
        } else {
            b = 0;
        }

        real a;
        if(y < N) {
            a = A[y * N + i];
        } else {
            a = 0;
        }
        
        // Get needed values from other threads.
        for(int j = 0; j < 32; j++){
            reg[j] += a * __shfl_sync(0xffffffff, b, j);
        }
    }

    // Load our row to C array.
    if(y < N) {
        for(int i = 0; i < 32; i++){
            if(x0 + i < N){
                C[y * N + y0 + i] = reg[i];
            }
        }
    }
}

void gpu_matrix_multiply_v5_32(real* A, real* B, real* C){
    real *A_gpu = NULL, *B_gpu = NULL, *C_gpu = NULL;
    make_gpu_matrices(A_gpu, B_gpu, C_gpu, A, B);
    dim3 threadsPerBlock(32, 1, 1);
    dim3 numBlocks(N / 32 + 1, N / 32 + 1, 1);
    kernel_v5_32<<<numBlocks, threadsPerBlock, 0>>>(A_gpu, B_gpu, C_gpu, N);
    delete_gpu_matrices(A_gpu, B_gpu, C_gpu, C);
}

void cpu_matrix_multiply(real* A, real* B, real* C){
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            C[i*N + j] = 0;
            for(int k = 0; k < N; k++){
                C[i*N + j] += A[i*N + k] * B[k*N + j];
            }
        }
    }
}
